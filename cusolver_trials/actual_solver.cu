#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>

#define BLOCK_SIZE 32

// Error checking macro for CUDA calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Error handling for cuSolver calls
inline void cusolveSafeCall(hipsolverStatus_t status) {
    if(status != HIPSOLVER_STATUS_SUCCESS) {
        std::cerr << "cuSolver API failed with status " << status << std::endl;
        exit(1);
    }
}

// Error handling for cuBLAS calls
inline void cublasSafeCall(hipblasStatus_t status) {
    if(status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS API failed with status " << status << std::endl;
        exit(1);
    }
}

__global__ void printMatrix(const double *A, int numRows, int numCols) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;

    if(row < numRows && col < numCols) {
        printf("A[%d, %d] = %f\n", row, col, A[row + col * numRows]);
    }
}

int main() {
//                    /\_[]_/\
//               |] _||_ [|
//        ___     \/ || \/
//       /___\       ||
//      (|0 0|)      ||
//    __/{\U/}\_ ___/vvv
//   / \  {~}   / _|_P|
//   | /\  ~   /_/   []
//   |_| (____)        
//   \_]/______\        -Defending the gates to the most flawlessly written cuSolver code-
//      _\_||_/_           
//     (_,_||_,_)

    const int Nrows = 3;
    const int Ncols = 3;

    //ASSUMPTION: Nrows >= Ncols 
    double h_A[Nrows * Ncols] = {1, 4, 7, 2, 5, 8, 3, 6, 10};  // Column-major storage
    double h_b[Nrows] = {1, 2, 3};

    // cuSOLVER and CUBLAS initialization
    hipsolverHandle_t solver_handle;
    hipblasHandle_t cublas_handle;
    cusolveSafeCall(hipsolverDnCreate(&solver_handle));
    cublasSafeCall(hipblasCreate(&cublas_handle));

    double *d_A, *d_TAU, *d_b, *work;
    int *devInfo, work_size = 0;

    gpuErrchk(hipMalloc((void**)&d_A, Nrows * Ncols * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&d_b, Nrows * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&d_TAU, Ncols * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&devInfo, sizeof(int)));

    gpuErrchk(hipMemcpy(d_A, h_A, Nrows * Ncols * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, h_b, Nrows * sizeof(double), hipMemcpyHostToDevice));

    cusolveSafeCall(hipsolverDnDgeqrf_bufferSize(solver_handle, Nrows, Ncols, d_A, Nrows, &work_size));
    gpuErrchk(hipMalloc((void**)&work, work_size * sizeof(double)));

    // QR decomposition 
    cusolveSafeCall(hipsolverDnDgeqrf(solver_handle, Nrows, Ncols, d_A, Nrows, d_TAU, work, work_size, devInfo));

    // Extracting the R matrix and compute Q^T*b
    double *d_R;
    gpuErrchk(hipMalloc(&d_R, Ncols * Ncols * sizeof(double))); // R is NxN (upper triangular)
    dim3 Grid(BLOCK_SIZE, BLOCK_SIZE);
    dim3 Block((Ncols + BLOCK_SIZE - 1) / BLOCK_SIZE, (Ncols + BLOCK_SIZE - 1) / BLOCK_SIZE);

    printMatrix<<<Block, Grid>>>(d_A, Nrows, Ncols);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // Computing Q^T*b (storing the result in d_b)
    cusolveSafeCall(hipsolverDnDormqr(solver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, Nrows, 1, Ncols, d_A, Nrows, d_TAU, d_b, Nrows, work, work_size, devInfo));

    // Solving R*x = Q^T*b for x
    const double alpha = 1.0;
    cublasSafeCall(hipblasDtrsm(cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, Ncols, 1, &alpha, d_A, Nrows, d_b, Nrows));

    // Copying the d_b solution into h_x for printing
    double h_x[Ncols];
    gpuErrchk(hipMemcpy(h_x, d_b, Ncols * sizeof(double), hipMemcpyDeviceToHost));

    // Printing to terminal x[i] solution
    std::cout << "Solution x: \n";
    for(int i = 0; i < Ncols; i++) {
        std::cout << "x[" << i << "] = " << h_x[i] << std::endl;
    }

    // DESTROY DEMOLISH DEVOUR 
    hipFree(d_A);
    hipFree(d_TAU);
    hipFree(d_b);
    hipFree(work);
    hipFree(devInfo);
    hipFree(d_R);

    hipsolverDnDestroy(solver_handle);
    hipblasDestroy(cublas_handle);

    return 0;
}
