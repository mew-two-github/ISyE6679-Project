#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>

// Error handling macros used by CUDA runtime methods
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Error handling for cuSolver operations
inline void cusolverSafeCall(hipsolverStatus_t status) {
    if(status != HIPSOLVER_STATUS_SUCCESS) {
        std::cerr << "cuSolver API failed with status " << status << std::endl;
        exit(1);
    }
}

// Error handling for cuBLAS operations
inline void cublasSafeCall(hipblasStatus_t status) {
    if(status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS API failed with status " << status << std::endl;
        exit(1);
    }
}

void solver( double* A, double* b, double* x, int Nrows, int Ncols) {
    hipsolverHandle_t solver_handle;
    hipblasHandle_t cublas_handle;
    cusolverSafeCall(hipsolverDnCreate(&solver_handle));
    cublasSafeCall(hipblasCreate(&cublas_handle));

    double *d_A, *d_TAU, *d_b, *work;
    int *devInfo, work_size = 0;

    gpuErrchk(hipMalloc((void**)&d_A, Nrows * Ncols * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&d_b, Nrows * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&d_TAU, Ncols * sizeof(double)));
    gpuErrchk(hipMalloc((void**)&devInfo, sizeof(int)));

    gpuErrchk(hipMemcpy(d_A, h_A.data(), Nrows * Ncols * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, h_b.data(), Nrows * sizeof(double), hipMemcpyHostToDevice));

    cusolverSafeCall(hipsolverDnDgeqrf_bufferSize(solver_handle, Nrows, Ncols, d_A, Nrows, &work_size));
    gpuErrchk(hipMalloc((void**)&work, work_size * sizeof(double)));

    cusolverSafeCall(hipsolverDnDgeqrf(solver_handle, Nrows, Ncols, d_A, Nrows, d_TAU, work, work_size, devInfo));
    cusolverSafeCall(hipsolverDnDormqr(solver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, Nrows, 1, Ncols, d_A, Nrows, d_TAU, d_b, Nrows, work, work_size, devInfo));

    const double alpha = 1.0;
    cublasSafeCall(hipblasDtrsm(cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, Ncols, 1, &alpha, d_A, Nrows, d_b, Nrows));

    std::vector<double> h_x(Ncols);
    gpuErrchk(hipMemcpy(h_x.data(), d_b, Ncols * sizeof(double), hipMemcpyDeviceToHost));

    hipFree(d_A);
    hipFree(d_TAU);
    hipFree(d_b);
    hipFree(work);
    hipFree(devInfo);

    hipsolverDnDestroy(solver_handle);
    hipblasDestroy(cublas_handle);
}

int main() {
    int m = 3, n = 3;
    double A[m*n] = {1, 2, 3, 4, 5, 6, 7, 8, 10}; 
    double b[m] = {1, 2, 3};
    double x[m];

    solveQR(A, b, m, n);

    std::cout << "Solution x: \n";
    for (int i = 0; i < result.size(); i++) {
        std::cout << "x[" << i << "] = " << result[i] << std::endl;
    }

    return 0;
}
