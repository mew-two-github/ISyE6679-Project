#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>

// Error handling macros
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#include <iostream>
#define CUSOLVER_CHECK(call) { \
    hipsolverStatus_t err; \
    if ((err = (call)) != HIPSOLVER_STATUS_SUCCESS) { \
        std::cerr << "CUSOLVER error: " << err << ", " << "file: " << __FILE__ << ", line: " << __LINE__ << std::endl; \
        exit(1); \
    } \
}

#define CUSPARSE_CHECK(call) \
    do { \
        hipsparseStatus_t error = call; \
        if (error != HIPSPARSE_STATUS_SUCCESS) { \
            printf("CUSPARSE error at %s:%d\n", __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

using namespace std;
void convertToCSR(double* matrix, int rows, int cols, int* row_ptr, int* col_idx, double* values, int nnz) 
{
    int i, j, k;

    
    nnz = 0;
    for (j = 0; j < cols; j++) {
        for (i = 0; i < rows; i++) {
            if (matrix[j * rows + i] != 0) {
                (nnz)++;
            }
        }
    }

    
    /*row_ptr = (int*)malloc((rows + 1) * sizeof(int));
    col_idx = (int*)malloc(nnz * sizeof(int));
    values = (double*)malloc(nnz * sizeof(double));*/

    
    k = 0;
    row_ptr[0] = 1;
    for (i = 0; i < rows; i++) 
    {
        for (j = 0; j < cols; j++) {
            if (matrix[j * rows + i] != 0) {
                values[k] = matrix[j * rows + i];
                col_idx[k] = j+1;
                k++;
            }
        }
        row_ptr[i + 1] = k;
    }
}

int main(int argc, char *argv[]) {
    hipsolverSpHandle_t cusolverH = NULL;
    csrqrInfo_t info = NULL;
    hipsparseMatDescr_t descrA = NULL;
    hipStream_t stream = NULL;

    int *d_csrRowPtrA = nullptr;
    int *d_csrColIndA = nullptr;
    double *d_csrValA = nullptr;
    double *d_b = nullptr; // batchSize * m
    double *d_x = nullptr; // batchSize * m

    size_t size_qr = 0;
    size_t size_internal = 0;
    void *buffer_qr = nullptr; // working space for numerical factorization


    const int m = 3;
    // const int nnzA = 7;
    int rows = m, cols = 3;
    double matrix[] = {1,4,7,2,5,8,3,6,9};
    int nnzA = 0;
        for (int j = 0; j < cols; j++) {
        for (int i = 0; i < rows; i++) {
            if (matrix[j * rows + i] != 0) {
                nnzA++;
            }
        }
    }
    int row_ptr[rows+1];
    int col_idx[nnzA];
    double values[nnzA];


    convertToCSR(matrix, rows, cols, row_ptr, col_idx, values, nnzA);

    std::vector<int> csrRowPtrAcpy(rows+1);
    std::vector<int> csrColIndAcpy(nnzA);
    std::vector<double> csrValAcpy(nnzA);
    // std::vector<double> b = {1,2,3};
    
    for(int i=0;i<rows+1;i++)    
    {
        csrRowPtrAcpy[i] = row_ptr[i];
	cout<<csrRowPtrAcpy[i]<<" ";
    }
    cout<<nnzA;
    for(int i =0;i<nnzA;i++)
    {
        csrValAcpy[i]= values[i];
        csrColIndAcpy[i] = col_idx[i];
	cout<<csrColIndAcpy[i]<<" ";
    }
    const std::vector<int> csrRowPtrA = csrRowPtrAcpy;
    const std::vector<int> csrColIndA = csrColIndAcpy;
    const std::vector<double> csrValA = csrValAcpy;
    const std::vector<double> b = {1,2,3};
    const int batchSize = 17;
    cout<<"line 126-130"<<endl;



    std::vector<double> csrValABatch(nnzA * batchSize);
    std::vector<double> bBatch(m * batchSize);
    std::vector<double> xBatch(m * batchSize);
    cout<<"line 134-138"<<endl;

    for (int colidx = 0; colidx < nnzA; colidx++) {
        double Areg = csrValA[colidx];
        for (int batchId = 0; batchId < batchSize; batchId++) {
            double eps = (static_cast<double>((std::rand() % 100) + 1)) * 1.e-4;
            csrValABatch[batchId * nnzA + colidx] = Areg + eps;
        }
    }
    cout<<"line 140-147"<<endl;

    for (int j = 0; j < m; j++) {
        double breg = b[j];
        for (int batchId = 0; batchId < batchSize; batchId++) {
            double eps = (static_cast<double>((std::rand() % 100) + 1)) * 1.e-4;
            bBatch[batchId * m + j] = breg + eps;
        }
    }
    cout<<"line 149-155"<<endl;

    CUSOLVER_CHECK(hipsolverSpCreate(&cusolverH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSpSetStream(cusolverH, stream));
    CUSPARSE_CHECK(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CHECK(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CUSPARSE_CHECK(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    CUSOLVER_CHECK(cusolverSpCreateCsrqrInfo(&info));
    cout<<"line 158-165"<<endl;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_csrValA), sizeof(double) * csrValABatch.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_csrColIndA), sizeof(int) * csrColIndA.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_csrRowPtrA), sizeof(int) * csrRowPtrA.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b), sizeof(double) * bBatch.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_x), sizeof(double) * xBatch.size()));
    cout<<"line 171"<<endl;
    CUDA_CHECK(hipMemcpyAsync(d_csrColIndA, csrColIndA.data(), sizeof(int) * csrColIndA.size(),
                               hipMemcpyHostToDevice, stream)); cout<<"line 174"<<endl;
    CUDA_CHECK(hipMemcpyAsync(d_csrRowPtrA, csrRowPtrA.data(), sizeof(int) * csrRowPtrA.size(),
                               hipMemcpyHostToDevice, stream));cout<<"line 175"<<endl;

    CUSOLVER_CHECK(cusolverSpXcsrqrAnalysisBatched(cusolverH, m, m, nnzA, descrA, d_csrRowPtrA,
                                                   d_csrColIndA, info));cout<<"line 178"<<endl;

    CUDA_CHECK(hipStreamSynchronize(stream)); cout<<"line 181"<<endl;
    cout<<"line 182"<<endl;

    size_t free_mem = 0;
    size_t total_mem = 0;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));

    int batchSizeMax = 2;
    while (batchSizeMax < batchSize) {
        CUSOLVER_CHECK(cusolverSpDcsrqrBufferInfoBatched(cusolverH, m, m, nnzA, descrA, d_csrValA, d_csrRowPtrA,
                                                         d_csrColIndA, batchSizeMax, info, &size_internal, &size_qr));
        if ((size_internal + size_qr) > free_mem) {
            batchSizeMax /= 2;
            break;
        }
        batchSizeMax *= 2;
    }
    cout<<"line 187-192"<<endl;
    batchSizeMax = std::min(batchSizeMax, batchSize);
    batchSizeMax = 2;

    CUSOLVER_CHECK(cusolverSpDcsrqrBufferInfoBatched(cusolverH, m, m, nnzA, descrA, d_csrValA, d_csrRowPtrA,
                                                     d_csrColIndA, batchSizeMax, info, &size_internal, &size_qr));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&buffer_qr), size_qr));
    cout<<"line 206"<<endl;

    for (int idx = 0; idx < batchSize; idx += batchSizeMax) {
        const int cur_batchSize = std::min(batchSizeMax, batchSize - idx);
        CUDA_CHECK(hipMemcpyAsync(d_csrValA, csrValABatch.data() + idx * nnzA,
                                   sizeof(double) * nnzA * cur_batchSize, hipMemcpyHostToDevice,
                                   stream));
        CUDA_CHECK(hipMemcpyAsync(d_b, bBatch.data() + idx * m, sizeof(double) * m * cur_batchSize,
                                   hipMemcpyHostToDevice, stream));

        CUSOLVER_CHECK(cusolverSpDcsrqrsvBatched(cusolverH, m, m, nnzA, descrA, d_csrValA,
                                                 d_csrRowPtrA, d_csrColIndA, d_b, d_x,
                                                 cur_batchSize, info, buffer_qr));

        CUDA_CHECK(hipMemcpyAsync(xBatch.data() + idx * m, d_x, sizeof(double) * m * cur_batchSize,
                                   hipMemcpyDeviceToHost, stream));
    }
    cout<<"line 223"<<endl;
    CUDA_CHECK(hipStreamSynchronize(stream));

    const int baseA = (HIPSPARSE_INDEX_BASE_ONE == hipsparseGetMatIndexBase(descrA)) ? 1 : 0;

    for (int batchId = 0; batchId < batchSize; batchId++) {
        double *csrValAj = csrValABatch.data() + batchId * nnzA;
        double *xj = xBatch.data() + batchId * m;
        double *bj = bBatch.data() + batchId * m;
        double sup_res = 0;
        for (int row = 0; row < m; row++) {
            const int start = csrRowPtrA[row] - baseA;
            const int end = csrRowPtrA[row + 1] - baseA;
            double Ax = 0.0;
            for (int colidx = start; colidx < end; colidx++) {
                const int col = csrColIndA[colidx] - baseA;
                const double Areg = csrValAj[colidx];
                const double xreg = xj[col];
                Ax += Areg * xreg;
            }
            double r = bj[row] - Ax;
            sup_res = std::max(sup_res, fabs(r));
        }
        printf("batchId %d: sup|bj - Aj*xj| = %E \n", batchId, sup_res);
    }

    for (int batchId = 0; batchId < batchSize; batchId++) {
        double *xj = xBatch.data() + batchId * m;
        for (int row = 0; row < m; row++) {
            printf("x%d[%d] = %E\n", batchId, row, xj[row]);
        }
        printf("\n");
    }

    CUDA_CHECK(hipFree(d_csrRowPtrA));
    CUDA_CHECK(hipFree(d_csrColIndA));
    CUDA_CHECK(hipFree(d_csrValA));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(buffer_qr));

    CUSOLVER_CHECK(hipsolverSpDestroy(cusolverH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
