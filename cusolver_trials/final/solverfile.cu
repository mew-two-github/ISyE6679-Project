#include <cstdio>
#include <cstdlib>
#include <vector>
#include<iostream>

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>

using namespace std;

// Error handling macros
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#include <iostream>
#define CUSOLVER_CHECK(call) { \
    hipsolverStatus_t err; \
    if ((err = (call)) != HIPSOLVER_STATUS_SUCCESS) { \
        std::cerr << "CUSOLVER error: " << err << ", " << "file: " << __FILE__ << ", line: " << __LINE__ << std::endl; \
        exit(1); \
    } \
}

#define CUSPARSE_CHECK(call) \
    do { \
        hipsparseStatus_t error = call; \
        if (error != HIPSPARSE_STATUS_SUCCESS) { \
            printf("CUSPARSE error at %s:%d\n", __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

void convertToCSR(double* matrix, int rows, int cols, int* row_ptr, int* col_idx, double* values, int nnz) 
{
    int i, j, k;
   
    /*row_ptr = (int*)malloc((rows + 1) * sizeof(int));
    col_idx = (int*)malloc(nnz * sizeof(int));
    values = (double*)malloc(nnz * sizeof(double));*/

    
    k = 0;
    row_ptr[0] = 1;
    for (i = 0; i < rows; i++) 
    {
        for (j = 0; j < cols; j++) {
            if (abs(matrix[j * rows + i]) <= 1e-8) {
                values[k] = matrix[j * rows + i];
                col_idx[k] = j+1;
                k++;
            }
        }
        row_ptr[i + 1] = k+1;
    }
}

extern "C" void solve(double* matrix,  double* b_arr, int m, double* res ) {
    hipsolverSpHandle_t cusolverH = NULL;
    csrqrInfo_t info = NULL;
    hipsparseMatDescr_t descrA = NULL;
    hipStream_t stream = NULL;

    int *d_csrRowPtrA = nullptr;
    int *d_csrColIndA = nullptr;
    double *d_csrValA = nullptr;
    double *d_b = nullptr; // batchSize * m
    double *d_x = nullptr; // batchSize * m

    size_t size_qr = 0;
    size_t size_internal = 0;
    void *buffer_qr = nullptr; // working space for numerical factorization
    // const int nnzA = 7;
    int rows = m, cols = m;
    
    int nnzA = 0;
        for (int j = 0; j < cols; j++) {
        for (int i = 0; i < rows; i++) {
            if (abs(matrix[j * rows + i]) <= 1e-8) {
                nnzA++;
            }
        }
    }
    int row_ptr[rows+1];
    int col_idx[nnzA];
    double values[nnzA];


    convertToCSR(matrix, rows, cols, row_ptr, col_idx, values, nnzA);

    std::vector<int> csrRowPtrAcpy(rows+1);
    std::vector<int> csrColIndAcpy(nnzA);
    std::vector<double> csrValAcpy(nnzA);
    // std::vector<double> b = {1,2,3};
    cout<<"Rowptrs\n";
    for(int i=0;i<rows+1;i++)    
    {
        csrRowPtrAcpy[i] = row_ptr[i];
    cout<<csrRowPtrAcpy[i]<<" ";
    }
    cout<<endl;
    cout<<"nnzA"<<nnzA;
    cout<<"\nCol Idx\n";
    for(int i =0;i<nnzA;i++)
    {
        csrValAcpy[i]= values[i];
        csrColIndAcpy[i] = col_idx[i];
        cout<<csrColIndAcpy[i]<<" ";
    }
    cout<<endl;
    const std::vector<int> csrRowPtrA = csrRowPtrAcpy;
    const std::vector<int> csrColIndA = csrColIndAcpy;
    const std::vector<double> csrValA = csrValAcpy;
    // const std::vector<double> b = {40,40,40,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,225.753,13.4936,-96.244,-54.9104,-7.6,5.38456,-2.84615e-311,24,-35.3885,-11.0896,-3.5,-8.85719,-14.7368,-14.9,25.3857,14.4312,-8.28304,29.5385,-1.66932,-1.52778,10.9658,-1.8,13.2174,-0.908676,-5,2.49465,2.51826,-5.01828e-310,0,0,0,0,0,0,0,0,0,0,0,0};
    vector<double> b(m);
    for(int i = 0;i < m;++i)
    {
        b[i] = b_arr[i];
    }
    const int batchSize = 1;
    cout<<"line 126-130"<<endl;


    std::vector<double> csrValABatch(nnzA * batchSize);
    std::vector<double> bBatch(m * batchSize);
    std::vector<double> xBatch(m * batchSize);

    for (int colidx = 0; colidx < nnzA; colidx++) {
        double Areg = csrValA[colidx];
        for (int batchId = 0; batchId < batchSize; batchId++) {
            double eps = (static_cast<double>((std::rand() % 100) + 1)) * 1.e-4;
            csrValABatch[batchId * nnzA + colidx] = Areg + eps;
        }
    }

    for (int j = 0; j < m; j++) {
        double breg = b[j];
        for (int batchId = 0; batchId < batchSize; batchId++) {
            double eps = (static_cast<double>((std::rand() % 100) + 1)) * 1.e-4;
            bBatch[batchId * m + j] = breg + eps;
        }
    }

    CUSOLVER_CHECK(hipsolverSpCreate(&cusolverH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSpSetStream(cusolverH, stream));
    CUSPARSE_CHECK(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CHECK(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CUSPARSE_CHECK(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    CUSOLVER_CHECK(cusolverSpCreateCsrqrInfo(&info));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_csrValA), sizeof(double) * csrValABatch.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_csrColIndA), sizeof(int) * csrColIndA.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_csrRowPtrA), sizeof(int) * csrRowPtrA.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b), sizeof(double) * bBatch.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_x), sizeof(double) * xBatch.size()));

    CUDA_CHECK(hipMemcpyAsync(d_csrColIndA, csrColIndA.data(), sizeof(int) * csrColIndA.size(),
                               hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_csrRowPtrA, csrRowPtrA.data(), sizeof(int) * csrRowPtrA.size(),
                               hipMemcpyHostToDevice, stream));

    CUSOLVER_CHECK(cusolverSpXcsrqrAnalysisBatched(cusolverH, m, m, nnzA, descrA, d_csrRowPtrA,
                                                   d_csrColIndA, info));

    CUDA_CHECK(hipStreamSynchronize(stream));

    size_t free_mem = 0;
    size_t total_mem = 0;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));

    int batchSizeMax = 2;
    while (batchSizeMax < batchSize) {
        CUSOLVER_CHECK(cusolverSpDcsrqrBufferInfoBatched(cusolverH, m, m, nnzA, descrA, d_csrValA, d_csrRowPtrA,
                                                         d_csrColIndA, batchSizeMax, info, &size_internal, &size_qr));
        if ((size_internal + size_qr) > free_mem) {
            batchSizeMax /= 2;
            break;
        }
        batchSizeMax *= 2;
    }
    cout<<"line 135"<<endl;
    batchSizeMax = std::min(batchSizeMax, batchSize);
    batchSizeMax = 2;

    CUSOLVER_CHECK(cusolverSpDcsrqrBufferInfoBatched(cusolverH, m, m, nnzA, descrA, d_csrValA, d_csrRowPtrA,
                                                     d_csrColIndA, batchSizeMax, info, &size_internal, &size_qr));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&buffer_qr), size_qr));
    cout<<"line 142"<<endl;
    for (int idx = 0; idx < batchSize; idx += batchSizeMax) {
        const int cur_batchSize = std::min(batchSizeMax, batchSize - idx);
        CUDA_CHECK(hipMemcpyAsync(d_csrValA, csrValABatch.data() + idx * nnzA,
                                   sizeof(double) * nnzA * cur_batchSize, hipMemcpyHostToDevice,
                                   stream));
        CUDA_CHECK(hipMemcpyAsync(d_b, bBatch.data() + idx * m, sizeof(double) * m * cur_batchSize,
                                   hipMemcpyHostToDevice, stream));

        CUSOLVER_CHECK(cusolverSpDcsrqrsvBatched(cusolverH, m, m, nnzA, descrA, d_csrValA,
                                                 d_csrRowPtrA, d_csrColIndA, d_b, d_x,
                                                 cur_batchSize, info, buffer_qr));

        CUDA_CHECK(hipMemcpyAsync(xBatch.data() + idx * m, d_x, sizeof(double) * m * cur_batchSize,
                                   hipMemcpyDeviceToHost, stream));
    }

    CUDA_CHECK(hipStreamSynchronize(stream));

    const int baseA = (HIPSPARSE_INDEX_BASE_ONE == hipsparseGetMatIndexBase(descrA)) ? 1 : 0;

    for (int batchId = 0; batchId < batchSize; batchId++) {
        double *csrValAj = csrValABatch.data() + batchId * nnzA;
        double *xj = xBatch.data() + batchId * m;
        double *bj = bBatch.data() + batchId * m;
        double sup_res = 0;
        for (int row = 0; row < m; row++) {
            const int start = csrRowPtrA[row] - baseA;
            const int end = csrRowPtrA[row + 1] - baseA;
            double Ax = 0.0;
            for (int colidx = start; colidx < end; colidx++) {
                const int col = csrColIndA[colidx] - baseA;
                const double Areg = csrValAj[colidx];
                const double xreg = xj[col];
                Ax += Areg * xreg;
            }
            double r = bj[row] - Ax;
            sup_res = std::max(sup_res, fabs(r));
        }
        printf("batchId %d: sup|bj - Aj*xj| = %E \n", batchId, sup_res);
    }

    for (int batchId = 0; batchId < batchSize; batchId++) {
        double *xj = xBatch.data() + batchId * m;
        for (int row = 0; row < m; row++) {
            printf("x%d[%d] = %E\n", batchId, row, xj[row]);
        }
        printf("\n");
    }
    double *xj = xBatch.data();
    for (int row = 0; row < m; row++) {
            res[row] = xj[row];
        }  

    CUDA_CHECK(hipFree(d_csrRowPtrA));
    CUDA_CHECK(hipFree(d_csrColIndA));
    CUDA_CHECK(hipFree(d_csrValA));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(buffer_qr));

    CUSOLVER_CHECK(hipsolverSpDestroy(cusolverH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());
}

