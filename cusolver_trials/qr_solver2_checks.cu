#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>

// Error checking for CUDA calls
#define CHECK_CUDA(call) {\
    const hipError_t error = call;\
    if (error != hipSuccess) {\
        printf("Error: %s:%d, ", __FILE__, __LINE__);\
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
        exit(1);\
    }\
}

// Error checking for cuSolver calls
#define CHECK_CUSOLVER(call, handle) {\
    const hipsolverStatus_t error = call;\
    if (error != HIPSOLVER_STATUS_SUCCESS) {\
        if (handle) hipsolverDnDestroy(handle);\
        printf("Error: %s:%d, ", __FILE__, __LINE__);\
        printf("CUSOLVER error code:%d\n", error);\
        exit(1);\
    }\
}

// Error checking for cuBLAS calls
#define CHECK_CUBLAS(call, handle) {\
    const hipblasStatus_t error = call;\
    if (error != HIPBLAS_STATUS_SUCCESS) {\
        if (handle) hipblasDestroy(handle);\
        printf("Error: %s:%d, ", __FILE__, __LINE__);\
        printf("CUBLAS error code:%d\n", error);\
        exit(1);\
    }\
}

int main() {
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    const int m = 3; // Number of rows of A
    const int n = 3; // Number of columns of A
    double A[m*n] = {1, 2, 3, 4, 5, 6, 7, 8, 10}; 
    double b[m] = {1, 2, 3}; 
    double *d_A = NULL, *d_tau = NULL, *d_b = NULL;
    int *devInfo = NULL;
    double *d_work = NULL;
    int lwork = 0;
    int info_gpu = 0;
    double alpha = 1.0;

    
    CHECK_CUDA(hipSetDevice(0)); 
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH), cusolverH);
    CHECK_CUBLAS(hipblasCreate(&cublasH), cublasH);

    // Allocate memory 
    CHECK_CUDA(hipMalloc((void**)&d_A, sizeof(double) * m * n));
    CHECK_CUDA(hipMalloc((void**)&d_tau, sizeof(double) * n));
    CHECK_CUDA(hipMalloc((void**)&d_b, sizeof(double) * m));
    CHECK_CUDA(hipMalloc((void**)&devInfo, sizeof(int)));

    // Copy host memory
    CHECK_CUDA(hipMemcpy(d_A, A, sizeof(double) * m * n, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, b, sizeof(double) * m, hipMemcpyHostToDevice));

    //space of geqrf
    CHECK_CUSOLVER(hipsolverDnDgeqrf_bufferSize(cusolverH, m, n, d_A, m, &lwork), cusolverH);
    CHECK_CUDA(hipMalloc((void**)&d_work, sizeof(double) * lwork));

    // QR factorization
    CHECK_CUSOLVER(hipsolverDnDgeqrf(cusolverH, m, n, d_A, m, d_tau, d_work, lwork, devInfo), cusolverH);
    CHECK_CUDA(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    if (info_gpu != 0) {
        printf("QR factorization failed, info: %d\n", info_gpu);
    }

    // apply Q^T to b
    CHECK_CUSOLVER(hipsolverDnDormqr(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, m, 1, n, d_A, m, d_tau, d_b, m, d_work, lwork, devInfo), cusolverH);
    CHECK_CUDA(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    if (info_gpu != 0) {
        printf("Application of Q^T to b failed, info: %d\n", info_gpu);
    }

    // solve R*x = Q^T*b
    CHECK_CUBLAS(hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, 1, &alpha, d_A, m, d_b, n), cublasH);

    // Copy result back 
    CHECK_CUDA(hipMemcpy(b, d_b, sizeof(double) * n, hipMemcpyDeviceToHost));

    
    printf("Solution:\n");
    for (int i = 0; i < n; i++) {
        printf("%f\n", b[i]);
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_tau);
    hipFree(d_b);
    hipFree(devInfo);
    hipFree(d_work);
    hipsolverDnDestroy(cusolverH);
    hipblasDestroy(cublasH);

    return 0;
}
